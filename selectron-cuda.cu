#include "hip/hip_runtime.h"
// CUDA Selectron prototype
//
// Patrick Walton <pcwalton@mozilla.com>
//
// Copyright (c) 2014 Mozilla Corporation

#include "selectron.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

__device__ uint32_t css_rule_hash_device(uint32_t key, uint32_t seed) {
    CSS_RULE_HASH(key, seed);
}

__device__ const css_rule *__restrict__ css_cuckoo_hash_find_precomputed_device(
        const css_cuckoo_hash *__restrict__ hash,
        int32_t key,
        int32_t left_index,
        int32_t right_index) {
    CSS_CUCKOO_HASH_FIND_PRECOMPUTED(hash, key, left_index, right_index);
}

__device__ const css_rule *__restrict__ css_cuckoo_hash_find_device(
        const css_cuckoo_hash *__restrict__ hash,
        int32_t key) {
    CSS_CUCKOO_HASH_FIND(hash, key, css_rule_hash_device);
}

__global__ void match_selectors_device(dom_node *first,
                                       const css_stylesheet *__restrict__ stylesheet) {
#if 0
    MATCH_SELECTORS(first,
                    stylesheet,
                    blockIdx.x * THREAD_COUNT + threadIdx.x,
                    css_cuckoo_hash_find_device);
#endif
    MATCH_SELECTORS_PRECOMPUTED(first,
                                stylesheet,
                                blockIdx.x * THREAD_COUNT + threadIdx.x,
                                css_cuckoo_hash_find_precomputed_device,
                                css_rule_hash_device);
}

// Main routine

int get_cuda_device(bool cpu) {
    int device_count;
    checkCudaErrors(hipGetDeviceCount(&device_count));

    for (int device = 0; device < device_count; device++) {
        hipDeviceProp_t device_props;
        hipGetDeviceProperties(&device_props, device);
        if (device_props.computeMode == hipComputeModeProhibited)
            continue;
        fprintf(stderr, "found device: %s\n", device_props.name);
        return device;
    }

    fprintf(stderr, "no device found\n");
    return 1;
}

int main(int argc, char **argv) {
    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    int device_id = get_cuda_device(false);
    checkCudaErrors(hipSetDevice(device_id));

    srand(time(NULL));

    // Create the rule tree on the host.
    css_stylesheet *host_stylesheet = (css_stylesheet *)malloc(sizeof(css_stylesheet));
    create_stylesheet(host_stylesheet);

    // Create the DOM tree on the host.
    int global_count = 0;
    dom_node *host_dom = (dom_node *)malloc(sizeof(struct dom_node) * NODE_COUNT);
    create_dom(host_dom, NULL, &global_count, 0);

    // Allocate the DOM tree and copy over.
    dom_node *device_dom;
    checkCudaErrors(hipMalloc((void **)&device_dom, sizeof(struct dom_node) * NODE_COUNT));
    dom_node *device_dom_host_mirror = (dom_node *)malloc(sizeof(struct dom_node) * NODE_COUNT);
    memcpy(device_dom_host_mirror, host_dom, sizeof(struct dom_node) * NODE_COUNT);
    munge_dom_pointers(device_dom_host_mirror,
                       (ptrdiff_t)((ptrdiff_t)device_dom_host_mirror - (ptrdiff_t)device_dom));
    checkCudaErrors(hipMemcpy(device_dom,
                               device_dom_host_mirror,
                               sizeof(struct dom_node) * NODE_COUNT,
                               hipMemcpyHostToDevice));

    // Allocate the rule tree and copy over.
    css_stylesheet *device_stylesheet;
    checkCudaErrors(hipMalloc((void **)&device_stylesheet, sizeof(css_stylesheet)));
    checkCudaErrors(hipMemcpy(device_stylesheet,
                               host_stylesheet,
                               sizeof(css_stylesheet),
                               hipMemcpyHostToDevice));

    // Create start/stop events.
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    // Execute the kernel on the GPU.
    checkCudaErrors(hipEventRecord(start));
    match_selectors_device<<<NODE_COUNT / THREAD_COUNT, THREAD_COUNT>>>(device_dom,
                                                                        device_stylesheet);
    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    float gpu_elapsed = 0.0f;
    checkCudaErrors(hipEventElapsedTime(&gpu_elapsed, start, stop));

    // Execute the kernel on the CPU.
    uint64_t cpu_start = mach_absolute_time();
    for (int i = 0; i < NODE_COUNT; i++) {
        match_selectors(host_dom, host_stylesheet, i);
    }
    float cpu_elapsed = (double)(mach_absolute_time() - cpu_start) / 1000000.0;

    fprintf(stderr,
            "Selector matching: GPU %g ms, CPU %g ms (parallel CPU estimate %g ms)\n",
            (double)gpu_elapsed,
            (double)cpu_elapsed,
            (double)cpu_elapsed / ESTIMATED_PARALLEL_SPEEDUP);

    // Do frame construction.
    cpu_start = mach_absolute_time();
    for (int i = 0; i < NODE_COUNT; i++) {
        create_frame(host_dom, i);
    }
    float frame_construction_cpu_elapsed = (double)(mach_absolute_time() - cpu_start) / 1000000.0;

    fprintf(stderr,
            "Frame construction: CPU %g ms (parallel CPU estimate %g ms)\n",
            (double)frame_construction_cpu_elapsed,
            (double)frame_construction_cpu_elapsed / ESTIMATED_PARALLEL_SPEEDUP);

    uint64_t total_cpu_elapsed = cpu_elapsed + frame_construction_cpu_elapsed;
    fprintf(stderr,
            "Total CPU: %g ms (parallel CPU estimate %g ms)\n",
            (double)total_cpu_elapsed,
            (double)total_cpu_elapsed / ESTIMATED_PARALLEL_SPEEDUP);

    float best_case_elapsed = fmax(gpu_elapsed, frame_construction_cpu_elapsed);
    float best_case_parallel_elapsed = fmax(
            (double)gpu_elapsed,
            frame_construction_cpu_elapsed / ESTIMATED_PARALLEL_SPEEDUP);
    fprintf(stderr,
            "Best-case: %g ms (parallel estimate %g ms)\n",
            (double)best_case_elapsed,
            (double)best_case_parallel_elapsed);

    checkCudaErrors(hipMemcpy(device_dom_host_mirror,
                               device_dom,
                               sizeof(struct dom_node) * NODE_COUNT,
                               hipMemcpyDeviceToHost));

    check_dom(device_dom_host_mirror);

    checkCudaErrors(hipFree(device_stylesheet));
    checkCudaErrors(hipFree(device_dom));

    hipDeviceReset();
    return 0;
}

