#include "hip/hip_runtime.h"
// CUDA Selectron prototype
//
// Patrick Walton <pcwalton@mozilla.com>
//
// Copyright (c) 2014 Mozilla Corporation

#include <mach/mach.h>
#include <mach/mach_time.h>

#include <stdbool.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define NODE_COUNT 131072
#define THREAD_COUNT 1024
#define MAX_DOM_DEPTH 10

#define ESTIMATED_PARALLEL_SPEEDUP  2.7

#define RULE_ID_MAX 25
#define NODE_ID_MAX 50
#define RULE_TAG_NAME_MAX 8
#define NODE_TAG_NAME_MAX 12

#define CSS_SELECTOR_TYPE_NONE      0
#define CSS_SELECTOR_TYPE_ID        1
#define CSS_SELECTOR_TYPE_TAG_NAME  2

#define HASH_SIZE   256

#ifdef MAX
#undef MAX
#endif
#define MAX(a,b)    ((a) > (b) ? (a) : (b))

#define LEFT_SEED   10
#define RIGHT_SEED  20

// FIXME(pcwalton): This is not really implemented properly; it should resize the table.

struct css_rule {
    int32_t type;
    int32_t value;
};

struct css_cuckoo_hash {
    int32_t left_seed;
    int32_t right_seed;
    css_rule left[HASH_SIZE];
    css_rule right[HASH_SIZE];
};

#define CSS_RULE_HASH(key, seed) \
    do {\
        uint32_t hash = 2166136261; \
        hash = hash ^ seed; \
        hash = hash * 16777619; \
        hash = hash ^ key; \
        hash = hash * 16777619; \
        return hash; \
    } while(0)

uint32_t css_rule_hash(uint32_t key, uint32_t seed) {
    CSS_RULE_HASH(key, seed);
}

__device__ uint32_t css_rule_hash_device(uint32_t key, uint32_t seed) {
    CSS_RULE_HASH(key, seed);
}

void css_cuckoo_hash_reset(css_cuckoo_hash *hash) {
    for (int i = 0; i < HASH_SIZE; i++) {
        hash->left[i].type = 0;
        hash->right[i].type = 0;
    }
}

void css_cuckoo_hash_reseed(css_cuckoo_hash *hash) {
    hash->left_seed = rand();
    hash->right_seed = rand();
}

void css_cuckoo_hash_init(css_cuckoo_hash *hash) {
    css_cuckoo_hash_reset(hash);
    css_cuckoo_hash_reseed(hash);
}

void css_cuckoo_hash_rehash(css_cuckoo_hash *hash) {
    fprintf(stderr, "rehash unimplemented\n");
    abort();
}

bool css_cuckoo_hash_insert_internal(css_cuckoo_hash *hash, css_rule *rule, bool right) {
    int hashval = css_rule_hash(rule->value, right ? RIGHT_SEED : LEFT_SEED);
    int index = hashval % HASH_SIZE;
    css_rule *list = right ? hash->right : hash->left;
    if (list[index].type != 0) {
        if (!css_cuckoo_hash_insert_internal(hash, &list[index], !right))
            return false;
    }

    list[index] = *rule;
    return true;
}

void css_cuckoo_hash_insert(css_cuckoo_hash *hash, css_rule *rule) {
    if (css_cuckoo_hash_insert_internal(hash, rule, false))
        return;
    css_cuckoo_hash_reseed(hash);
    css_cuckoo_hash_rehash(hash);
    if (css_cuckoo_hash_insert_internal(hash, rule, false))
        return;
    fprintf(stderr, "rehashing failed\n");
    abort();
}

#define CSS_CUCKOO_HASH_FIND(hash, key, hashfn) \
    do {\
        int left_index = hashfn(key, LEFT_SEED) % HASH_SIZE; \
        if (hash->left[left_index].type != 0 && hash->left[left_index].value == key) \
            return &hash->left[left_index]; \
        int right_index = hashfn(key, RIGHT_SEED) % HASH_SIZE; \
        if (hash->right[right_index].type != 0 && hash->right[right_index].value == key) \
            return &hash->right[right_index]; \
        return NULL; \
    } while(0)

#define CSS_CUCKOO_HASH_FIND_PRECOMPUTED(hash, key, left_index, right_index) \
    do {\
        if (hash->left[left_index].type != 0 && hash->left[left_index].value == key) \
            return &hash->left[left_index]; \
        if (hash->right[right_index].type != 0 && hash->right[right_index].value == key) \
            return &hash->right[right_index]; \
        return NULL; \
    } while(0)

__device__ const css_rule *__restrict__ css_cuckoo_hash_find_device(
        const css_cuckoo_hash *__restrict__ hash,
        int32_t key) {
    CSS_CUCKOO_HASH_FIND(hash, key, css_rule_hash_device);
}

css_rule *css_cuckoo_hash_find(css_cuckoo_hash *hash, int32_t key) {
    CSS_CUCKOO_HASH_FIND(hash, key, css_rule_hash);
}

__device__ const css_rule *__restrict__ css_cuckoo_hash_find_precomputed_device(
        const css_cuckoo_hash *__restrict__ hash,
        int32_t key,
        int32_t left_index,
        int32_t right_index) {
    CSS_CUCKOO_HASH_FIND_PRECOMPUTED(hash, key, left_index, right_index);
}

css_rule *css_cuckoo_hash_find_precomputed(css_cuckoo_hash *hash,
                                           int32_t key,
                                           int32_t left_index,
                                           int32_t right_index) {
    CSS_CUCKOO_HASH_FIND_PRECOMPUTED(hash, key, left_index, right_index);
}

struct css_stylesheet_source {
    css_cuckoo_hash ids;
    css_cuckoo_hash tag_names;
};

struct css_stylesheet {
    css_stylesheet_source author;
    css_stylesheet_source user_agent;
};

struct dom_node {
    struct dom_node *parent;
    struct dom_node *first_child;
    struct dom_node *last_child;
    struct dom_node *next_sibling;
    struct dom_node *prev_sibling;
    int32_t id;
    int32_t tag_name;
    int32_t applicable_declaration_count;
    struct css_rule applicable_declarations[16];
};

#define MATCH_SELECTORS_HASH(node, hash, findfn) \
    do {\
        const css_rule *__restrict__ rule = findfn(hash, node->id); \
        if (rule != NULL) \
            node->applicable_declarations[node->applicable_declaration_count++] = *rule; \
    } while(0)

#define MATCH_SELECTORS(first, stylesheet, index, findfn) \
    do {\
        dom_node *node = &first[index]; \
        node->applicable_declaration_count = 0; \
        MATCH_SELECTORS_HASH(node, &stylesheet->author.ids, findfn); \
        MATCH_SELECTORS_HASH(node, &stylesheet->author.tag_names, findfn); \
        MATCH_SELECTORS_HASH(node, &stylesheet->user_agent.ids, findfn); \
        MATCH_SELECTORS_HASH(node, &stylesheet->user_agent.tag_names, findfn); \
    } while(0)

#define MATCH_SELECTORS_HASH_PRECOMPUTED(node, hash, findfn, left_index, right_index) \
    do {\
        const css_rule *__restrict__ rule = findfn(hash, node->id, left_index, right_index); \
        if (rule != NULL) \
            node->applicable_declarations[node->applicable_declaration_count++] = *rule; \
    } while(0)

#define MATCH_SELECTORS_PRECOMPUTED(first, stylesheet, index, findfn, hashfn) \
    do {\
        dom_node *node = &first[index]; \
        node->applicable_declaration_count = 0; \
        int32_t left_id_index = hashfn(node->id, LEFT_SEED) % HASH_SIZE; \
        int32_t right_id_index = hashfn(node->id, RIGHT_SEED) % HASH_SIZE; \
        int32_t left_tag_name_index = hashfn(node->tag_name, LEFT_SEED) % HASH_SIZE; \
        int32_t right_tag_name_index = hashfn(node->tag_name, RIGHT_SEED) % HASH_SIZE; \
        MATCH_SELECTORS_HASH_PRECOMPUTED(node, \
                                         &stylesheet->author.ids, \
                                         findfn, \
                                         left_id_index, \
                                         right_id_index); \
        MATCH_SELECTORS_HASH_PRECOMPUTED(node, \
                                         &stylesheet->author.tag_names, \
                                         findfn, \
                                         left_tag_name_index, \
                                         right_tag_name_index); \
        MATCH_SELECTORS_HASH_PRECOMPUTED(node, \
                                         &stylesheet->user_agent.ids, \
                                         findfn, \
                                         left_id_index, \
                                         right_id_index); \
        MATCH_SELECTORS_HASH_PRECOMPUTED(node, \
                                         &stylesheet->user_agent.tag_names, \
                                         findfn, \
                                         left_tag_name_index, \
                                         right_tag_name_index); \
    } while(0)

__global__ void match_selectors_device(dom_node *first,
                                       const css_stylesheet *__restrict__ stylesheet) {
#if 0
    MATCH_SELECTORS(first,
                    stylesheet,
                    blockIdx.x * THREAD_COUNT + threadIdx.x,
                    css_cuckoo_hash_find_device);
#endif
    MATCH_SELECTORS_PRECOMPUTED(first,
                                stylesheet,
                                blockIdx.x * THREAD_COUNT + threadIdx.x,
                                css_cuckoo_hash_find_precomputed_device,
                                css_rule_hash_device);
}

void match_selectors(dom_node *first, css_stylesheet *stylesheet, int32_t index) {
#if 0
    MATCH_SELECTORS(first, stylesheet, index, css_cuckoo_hash_find);
#endif
    MATCH_SELECTORS_PRECOMPUTED(first,
                                stylesheet,
                                index,
                                css_cuckoo_hash_find_precomputed,
                                css_rule_hash);
}

void create_stylesheet(css_stylesheet *stylesheet) {
    css_cuckoo_hash_init(&stylesheet->author.ids);
    css_cuckoo_hash_init(&stylesheet->author.tag_names);
    css_cuckoo_hash_init(&stylesheet->user_agent.ids);
    css_cuckoo_hash_init(&stylesheet->user_agent.tag_names);

    for (int i = 0; i < RULE_ID_MAX; i++) {
        css_rule rule = { CSS_SELECTOR_TYPE_ID, i };
        css_cuckoo_hash_insert(&stylesheet->author.ids, &rule);
    }
    for (int i = 0; i < RULE_ID_MAX; i++) {
        css_rule rule = { CSS_SELECTOR_TYPE_ID, i };
        css_cuckoo_hash_insert(&stylesheet->user_agent.ids, &rule);
    }
    for (int i = 0; i < RULE_TAG_NAME_MAX; i++) {
        css_rule rule = { CSS_SELECTOR_TYPE_TAG_NAME, i };
        css_cuckoo_hash_insert(&stylesheet->author.tag_names, &rule);
    }
    for (int i = 0; i < RULE_TAG_NAME_MAX; i++) {
        css_rule rule = { CSS_SELECTOR_TYPE_TAG_NAME, i };
        css_cuckoo_hash_insert(&stylesheet->user_agent.tag_names, &rule);
    }
}

void create_dom(dom_node *dest, dom_node *parent, int *global_count, int depth) {
    if (*global_count == NODE_COUNT)
        return;
    if (depth == MAX_DOM_DEPTH)
        return;

    dom_node *node = &dest[(*global_count)++];
    node->id = rand() % NODE_ID_MAX;
    node->tag_name = rand() % NODE_TAG_NAME_MAX;
    node->applicable_declaration_count = 0;

    node->first_child = node->last_child = node->next_sibling = NULL;
    if ((node->parent = parent) != NULL) {
        if (node->parent->last_child != NULL) {
            node->prev_sibling = node->parent->last_child;
            node->prev_sibling->next_sibling = node->parent->last_child = node;
        } else {
            node->parent->first_child = node->parent->last_child = node;
            node->prev_sibling = NULL;
        }
    }

    int child_count = rand() % (NODE_COUNT / 100);
    for (int i = 0; i < child_count; i++)
        create_dom(dest, node, global_count, depth + 1);
}

void munge_dom_pointers(dom_node *node, ptrdiff_t offset) {
    for (int i = 0; i < NODE_COUNT; i++) {
        node->parent = (dom_node *)((ptrdiff_t)node->parent + offset);
        node->first_child = (dom_node *)((ptrdiff_t)node->first_child + offset);
        node->last_child = (dom_node *)((ptrdiff_t)node->last_child + offset);
        node->next_sibling = (dom_node *)((ptrdiff_t)node->next_sibling + offset);
        node->prev_sibling = (dom_node *)((ptrdiff_t)node->prev_sibling + offset);
    }
}

void check_dom(dom_node *node) {
    for (int i = 0; i < 20; i++) {
        printf("%d -> %d\n", node[i].id, node[i].applicable_declaration_count);
    }
}

// Frame tree

struct frame {
    struct dom_node *node;
    int32_t type;
};

void create_frame(struct dom_node *first, int i) {
    struct frame *frame = (struct frame *)malloc(sizeof(struct frame));
    frame->node = &first[i];
    frame->type = 0;
}

// Main routine

int get_cuda_device(bool cpu) {
    int device_count;
    checkCudaErrors(hipGetDeviceCount(&device_count));

    for (int device = 0; device < device_count; device++) {
        hipDeviceProp_t device_props;
        hipGetDeviceProperties(&device_props, device);
        if (device_props.computeMode == hipComputeModeProhibited)
            continue;
        fprintf(stderr, "found device: %s\n", device_props.name);
        return device;
    }

    fprintf(stderr, "no device found\n");
    return 1;
}

int main(int argc, char **argv) {
    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    int device_id = get_cuda_device(false);
    checkCudaErrors(hipSetDevice(device_id));

    srand(time(NULL));

    // Create the rule tree on the host.
    css_stylesheet *host_stylesheet = (css_stylesheet *)malloc(sizeof(css_stylesheet));
    create_stylesheet(host_stylesheet);

    // Create the DOM tree on the host.
    int global_count = 0;
    dom_node *host_dom = (dom_node *)malloc(sizeof(struct dom_node) * NODE_COUNT);
    create_dom(host_dom, NULL, &global_count, 0);

    // Allocate the DOM tree and copy over.
    dom_node *device_dom;
    checkCudaErrors(hipMalloc((void **)&device_dom, sizeof(struct dom_node) * NODE_COUNT));
    dom_node *device_dom_host_mirror = (dom_node *)malloc(sizeof(struct dom_node) * NODE_COUNT);
    memcpy(device_dom_host_mirror, host_dom, sizeof(struct dom_node) * NODE_COUNT);
    munge_dom_pointers(device_dom_host_mirror,
                       (ptrdiff_t)((ptrdiff_t)device_dom_host_mirror - (ptrdiff_t)device_dom));
    checkCudaErrors(hipMemcpy(device_dom,
                               device_dom_host_mirror,
                               sizeof(struct dom_node) * NODE_COUNT,
                               hipMemcpyHostToDevice));

    // Allocate the rule tree and copy over.
    css_stylesheet *device_stylesheet;
    checkCudaErrors(hipMalloc((void **)&device_stylesheet, sizeof(css_stylesheet)));
    checkCudaErrors(hipMemcpy(device_stylesheet,
                               host_stylesheet,
                               sizeof(css_stylesheet),
                               hipMemcpyHostToDevice));

    // Create start/stop events.
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    // Execute the kernel on the GPU.
    checkCudaErrors(hipEventRecord(start));
    match_selectors_device<<<NODE_COUNT / THREAD_COUNT, THREAD_COUNT>>>(device_dom,
                                                                        device_stylesheet);
    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    float gpu_elapsed = 0.0f;
    checkCudaErrors(hipEventElapsedTime(&gpu_elapsed, start, stop));

    // Execute the kernel on the CPU.
    uint64_t cpu_start = mach_absolute_time();
    for (int i = 0; i < NODE_COUNT; i++) {
        match_selectors(host_dom, host_stylesheet, i);
    }
    float cpu_elapsed = (double)(mach_absolute_time() - cpu_start) / 1000000.0;

    fprintf(stderr,
            "Selector matching: GPU %g ms, CPU %g ms (parallel CPU estimate %g ms)\n",
            (double)gpu_elapsed,
            (double)cpu_elapsed,
            (double)cpu_elapsed / ESTIMATED_PARALLEL_SPEEDUP);

    // Do frame construction.
    cpu_start = mach_absolute_time();
    for (int i = 0; i < NODE_COUNT; i++) {
        create_frame(host_dom, i);
    }
    float frame_construction_cpu_elapsed = (double)(mach_absolute_time() - cpu_start) / 1000000.0;

    fprintf(stderr,
            "Frame construction: CPU %g ms (parallel CPU estimate %g ms)\n",
            (double)frame_construction_cpu_elapsed,
            (double)frame_construction_cpu_elapsed / ESTIMATED_PARALLEL_SPEEDUP);

    uint64_t total_cpu_elapsed = cpu_elapsed + frame_construction_cpu_elapsed;
    fprintf(stderr,
            "Total CPU: %g ms (parallel CPU estimate %g ms)\n",
            (double)total_cpu_elapsed,
            (double)total_cpu_elapsed / ESTIMATED_PARALLEL_SPEEDUP);

    float best_case_elapsed = fmax(gpu_elapsed, frame_construction_cpu_elapsed);
    float best_case_parallel_elapsed = fmax(
            (double)gpu_elapsed,
            frame_construction_cpu_elapsed / ESTIMATED_PARALLEL_SPEEDUP);
    fprintf(stderr,
            "Best-case: %g ms (parallel estimate %g ms)\n",
            (double)best_case_elapsed,
            (double)best_case_parallel_elapsed);

    checkCudaErrors(hipMemcpy(device_dom_host_mirror,
                               device_dom,
                               sizeof(struct dom_node) * NODE_COUNT,
                               hipMemcpyDeviceToHost));

    check_dom(device_dom_host_mirror);

    checkCudaErrors(hipFree(device_stylesheet));
    checkCudaErrors(hipFree(device_dom));

    hipDeviceReset();
    return 0;
}

